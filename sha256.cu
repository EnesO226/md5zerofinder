#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>
#include <string.h>
#include <stdint.h>
#include <cstdint>

__constant__ uint32_t K[64] = 
{   0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 
};

__device__ uint32_t s0(uint32_t x) {
    return ((x >> 7) | (x << (32 - 7))) ^ ((x >> 18) | (x << (32 - 18))) ^ (x >> 3);
}

__device__ uint32_t s1(uint32_t x) {
    return ((x >> 17) | (x << (32 - 17))) ^ ((x >> 19) | (x << (32 - 19))) ^ (x >> 10);
}

__device__ uint32_t S0(uint32_t x) {
    return ((x >> 2) | (x << (32 - 2))) ^ ((x >> 13) | (x << (32 - 13))) ^ ((x >> 22) | (x << (32 - 22)));
}

__device__ uint32_t S1(uint32_t x) {
    return ((x >> 6) | (x << (32 - 6))) ^ ((x >> 11) | (x << (32 - 11))) ^ ((x >> 25) | (x << (32 - 25)));
}

__device__ uint32_t maj(uint32_t a, uint32_t b, uint32_t c) {
    return (a & b) ^ (a & c) ^ (b & c);
}

__device__ uint32_t ch(uint32_t e, uint32_t f, uint32_t g) {
    return (e & f) ^ (~e &  g);
}

__device__ void transform(uint32_t state[], uint32_t block[]) {

    uint32_t a = 0x6a09e667;
    uint32_t b = 0xbb67ae85;
    uint32_t c = 0x3c6ef372;
    uint32_t d = 0xa54ff53a;
    uint32_t e = 0x510e527f;
    uint32_t f = 0x9b05688c;
    uint32_t g = 0x1f83d9ab;
    uint32_t h = 0x5be0cd19;

    uint32_t a0 = 0x6a09e667;
    uint32_t b0 = 0xbb67ae85;
    uint32_t c0 = 0x3c6ef372;
    uint32_t d0 = 0xa54ff53a;
    uint32_t e0 = 0x510e527f;
    uint32_t f0 = 0x9b05688c;
    uint32_t g0 = 0x1f83d9ab;
    uint32_t h0 = 0x5be0cd19;

    uint32_t x[64];

    for (int i = 0; i < 16; i++) {
        x[i] = block[i];
    }

    for (int j = 16; j < 64; j++) {
        x[j] = x[j - 16] + s0(x[j - 15]) + x[j - 7] + s1(x[j - 2]);
    }

    for (int k = 0; k < 64; k++) {
        uint32_t sig1 = S1(e);
        uint32_t choose = ch(e, f, g);
        uint32_t temp1 = h + sig1 + choose + K[k] + x[k];
        uint32_t sig0 = S0(a);
        uint32_t majority = maj(a, b, c);
        uint32_t temp2 = sig0 + majority;

        h = g;
        g = f;
        f = e;
        e = d + temp1;
        d = c;
        c = b;
        b = a;
        a = temp1 + temp2;
    }

    a0 += a;
    b0 += b;
    c0 += c;
    d0 += d;
    e0 += e;
    f0 += f;
    g0 += g;
    h0 += h;

    if (a0 == 0xbbee11aa && b0 < 0xffffffff) {
        printf("Compressed hash ---> %08x%08x%08x%08x%08x%08x%08x%08x\n", a0, b0, c0, d0, e0, f0, g0, h0);
        printf("For input block ---> %08x%08x\n\n", block[0], block[1]);
    }
}

__global__ void Test() {
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t msg[16];
    uint32_t state[4];
    for (int i = 0; i < 16; i++) {
        msg[i] = 0;
    }
    msg[0] = thread;
    
    for (uint64_t j = 0; j < 0xffffffffffffffff; j++) {
        msg[1] = (uint32_t)(j);
        msg[2] = 0x80000000;
        msg[15] = 0x00000040;
        transform(state, msg);
    }
}



int main()
{
    Test << <1024, 1024 >> > ();
    system("pause");
    return 0;
}